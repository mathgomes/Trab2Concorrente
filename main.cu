#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <math.h>
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/features2d/features2d.hpp"
#include <cstdio>

using namespace std;
using namespace cv;

//Tamanho da área de cada pedaço a ser feita a filtragem na imagem.
#define SQUARE_AREA 25

//Número de threads a serem utilizadas.
#define NUMTHREAD 1024

__device__ unsigned char smooth(unsigned char* pixel, int i, int j, int w, int h);
__global__ void filtro(unsigned char *in, unsigned char *out, int w, int h);

int main( int argc, char** argv) {

	int tam, tamLargura, tamAltura, blocoX , blocoY, i; 
  double tempo;
  Mat src, dst[3], finImg;
  clock_t itime, ftime;
	unsigned char *dados_in, *dados_out;

  int imgType = atoi(argv[2]);
  src = imread( argv[1], imgType );
	
	//obtém o tamanho da img de entrada.
	tamAltura = src.rows;
	tamLargura = src.cols;
	
	// Aloca vetores para separar a imagem.
	tam = tamLargura * tamAltura * sizeof(unsigned char);
	
	hipMalloc((void**)&dados_in, tam);
	hipMalloc((void**)&dados_out, tam);

	// Npumero de blocos para cada dimensão
	blocoX = (int)ceil((double) tamLargura/(double)NUMTHREAD);
	blocoY = tamAltura;

	// define o número de blocos e threads.
	dim3 Blocos(blocoX, blocoY);
	dim3 threadBloco(NUMTHREAD);
	
  itime = clock();
	
  /// grayScale image section
  if( imgType == 0) {
	
    //Passa o filtro no único canal da img em GRAY
		hipMemcpy(dados_in, (unsigned char*) src.data , tam,  hipMemcpyHostToDevice);
		filtro<<<Blocos, threadBloco>>>(dados_in, dados_out, tamLargura, tamAltura);
		hipMemcpy((unsigned char*) src.data , dados_out, tam, hipMemcpyDeviceToHost);
		imwrite("novaImg.jpg", src);
		
  }else{

	    /// Split the image in channels
	    split(src,dst);
	    
	    /// Apply medianBlur in each channel
	    for(int i=0;i<3;++i){
	    
			hipMemcpy(dados_in, (unsigned char*) dst[i].data , tam,  hipMemcpyHostToDevice);
			filtro<<<Blocos, threadBloco>>>(dados_in, dados_out, tamLargura, tamAltura);
			hipMemcpy((unsigned char*) dst[i].data , dados_out, tam, hipMemcpyDeviceToHost);
			
	    }

	    /// Push the channels into the Mat vector
	    vector<Mat> rgb;
	    rgb.push_back(dst[0]); //blue
	    rgb.push_back(dst[1]); //green
	    rgb.push_back(dst[2]); //red

	    /// Merge the three channels
	    merge(rgb, finImg);

	    imwrite("novaImg.jpg", finImg);
  }
    
  ftime = clock();
  tempo = (ftime-itime) / (CLOCKS_PER_SEC * 1.0);
  printf("\nTempo : %lf\n",tempo);
	
  hipFree(dados_in);
	hipFree(dados_out);
	
  return 0;
}

//Método Smooth para processamento de imagem.
__device__ unsigned char smooth(unsigned char* pixel, int i, int j, int w, int h){
	
	int l, k;
	int sum;
	int raio = 5/2;
	
	sum = 0;
	for(l = i - raio; l <= i + raio; l++) {
		for(k = j - raio; k <= j + raio; k++) {
			if(l >= 0 && k >= 0 && l < h && k < w) {
				sum += pixel[l*w + k];
			}
		}
	}

	return sum/SQUARE_AREA;
}


//método para obter o pixel para a filtragem.
__global__ void filtro(unsigned char *in, unsigned char *out, int w, int h) {
	
	int i, j;

	i = blockIdx.y;
	j = blockIdx.x*blockDim.x + threadIdx.x;

	out[i*w+j] = smooth(in, i, j, w, h);
}
